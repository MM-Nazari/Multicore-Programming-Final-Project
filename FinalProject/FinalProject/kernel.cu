#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>
#include "hip/hip_runtime.h"
#include ""

#define MAX_WORD_LENGTH 100
#define MAX_WORDS 7


hipError_t histogram(int *histogram, char words);


__global__ void createHistogram(const char* words, int* histogram, int numWords) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (int i = tid; i < numWords; i += stride) {
		atomicAdd(&histogram[words[i]], 1);
	}
}

void print_array(int* a) {
	int i;
	printf("[-] histogram: ");
	for (i = 0; i < NUM_CLASSES; ++i) {
		printf("%d, ", a[i]);
	}
	printf("\b\b  \n");
}


int main()
{
	char words[MAX_WORDS][MAX_WORD_LENGTH] = { "alex","bruce","calvin","daniel","ethan",
		"ford","gale" };
	int numWords = 0;

	char* histogram = (char*)malloc(n * sizeof(char*));

	size_t maxWordLength = 0;
	for (int i = 0; i < numWords; i++) {
		size_t len = strlen(words[i]);
		if (len > maxWordLength) {
			maxWordLength = len;
		}

	histogram(histogram, words, n);

	print_array(histogram);

    return 0;
}


hipError_t histogram(int *histogram, char words, int n)
{
	char* deviceWords;
	hipMalloc((void**)&deviceWords, numWords * MAX_WORD_LENGTH * sizeof(char));
	hipMemcpy(deviceWords, words, numWords * MAX_WORD_LENGTH * sizeof(char), hipMemcpyHostToDevice);

	int* deviceHistogram;
	hipMalloc((void**)&deviceHistogram, MAX_WORDS * sizeof(int));
	hipMemset(deviceHistogram, 0, MAX_WORDS * sizeof(int));

	int blockSize = 256;
	int gridSize = (numWords + blockSize - 1) / blockSize;
	createHistogram << <gridSize, blockSize >> > (deviceWords, deviceHistogram, numWords);

	int* hostHistogram = new int[MAX_WORDS];
	hipMemcpy(hostHistogram, deviceHistogram, MAX_WORDS * sizeof(int), hipMemcpyDeviceToHost);

	for (int i = 0; i < numWords; i++) {
		if (hostHistogram[words[i]] > 0) {
			printf("%s: %d\n", words[i], hostHistogram[words[i]]);
		}
	}

	hipFree(deviceWords);
	hipFree(deviceHistogram);
	delete[] hostHistogram;

    
    return cudaStatus;
}
